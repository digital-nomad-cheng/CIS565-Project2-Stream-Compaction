#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        #define BLOCK_SIZE 512
        // TODO: 
        /**
         * Koggle Stone **inclusive** parallel scan using double buffering.
         * 
         */ 
        __global__ void scan_ks_kernel(int n, int* dev_odata, const int* dev_idata) {
            // double buffering
            __shared__ int T0[BLOCK_SIZE];
            __shared__ int T1[BLOCK_SIZE];

            int bi = blockIdx.x;
            int ti = threadIdx.x;
            int index = bi * blockDim.x + ti;

            int *src = T0;
            int *dest = T1;

            if (index < n) {
                T0[ti] = dev_idata[index];
                T1[ti] = T0[ti];
            }

            for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
                __syncthreads();
                if (ti >= stride)
                    dest[ti] = src[ti] + src[ti - stride];
                else
                    dest[ti] = src[ti];
                int* tmp = src;
                src = dest;
                dest = tmp;
            }
            if (index < n) {
                dev_odata[index] = src[ti];
            }
        }

        /**
         * Addition on partial sum per block
         */
        __global__ void add(float* block_sums, float* input, int len) {
            int bi = blockIdx.x;
            int ti = threadIdx.x;
            int index = (bi + 1) * blockDim.x + ti;
            if (index < len) {
                input[index] += block_sums[bi];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_odata;
            int* dev_idata;
            hipMalloc((void**)&dev_idata, sizeof(int) * n);
            hipMalloc((void**)&dev_odata, sizeof(int) * n);
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            dim3 gridDim = { 1, 1, 1 };
            dim3 blockDim = { BLOCK_SIZE, 1, 1 };
            timer().startGpuTimer();
            // TODO
            scan_ks_kernel <<< gridDim, blockDim >>> (n, dev_odata, dev_idata);
            hipDeviceSynchronize();
            timer().endGpuTimer();
            // since the scan kernel is inclusive, we have to set the first element to 0
            hipMemcpy(odata+1, dev_odata, sizeof(int) * (n-1), hipMemcpyDeviceToHost);
            odata[0] = 0;
            checkCUDAError("navie scan");
        }
    }
}
